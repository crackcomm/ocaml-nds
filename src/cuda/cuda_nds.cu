#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <unistd.h>

extern "C" {
#ifdef MATLAB_MEX_FILE
    #include "mex.h"
#else
    #include "utils.h"
#endif
}

#include <hipcub/hipcub.hpp>

#define BUFSIZE 100
#define checkCuda(result) __checkCuda(result, __FILE__, __LINE__)
inline hipError_t __checkCuda(hipError_t result, const char *file, const int line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%i: %s\n", file, line, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

#define BLOCK_X   32
#define BLOCK_Y    8
#define BLOCK_1D 512

void send_help(char *program) {
    fprintf(stderr, "Usage: %s [options] <n> <m> <pop_file>\n", program);
    fprintf(stderr, "Where:\n");
    fprintf(stderr, "  n         Population size.\n");
    fprintf(stderr, "  m         Number of objectives.\n");
    fprintf(stderr, "  pop_file  Path to the file containing the population.\n\n");

    fprintf(stderr, "Optional parameters:\n");
    fprintf(stderr, "  -h  Show this help\n");
    fprintf(stderr, "  -v  Verbosity level: 0 (only nds runtime)\n");
    fprintf(stderr, "                       1 (default, only front counts)\n");
    fprintf(stderr, "                       2 (front counts and contents).\n");
}

double get_time() {
    struct timespec ts;
    if (clock_gettime(CLOCK_MONOTONIC, &ts) == 0)
        return ts.tv_sec * 1000 + ts.tv_nsec / 1000000.0;
    else
        return 0;
}

__global__ void init_indexes(int n, int m, int *indexes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n*m) {
        indexes[i] = i % n;
    }
}

__global__ void transpose(int nx, int ny, float *tr_pop, const float *pop) {
    __shared__ float tile[BLOCK_X][BLOCK_X+1];

    int i = blockIdx.x * BLOCK_X + threadIdx.x;
    int j = blockIdx.y * BLOCK_X + threadIdx.y;

    if (i < nx) {
        for (int k = 0; k < BLOCK_X; k+= BLOCK_Y) {
            if ((j + k) >= ny) {
                break;
            }
            tile[threadIdx.y + k][threadIdx.x] = pop[(j + k)*nx + i];
        }
    }

    __syncthreads();

    i = blockIdx.y * BLOCK_X + threadIdx.x;
    j = blockIdx.x * BLOCK_X + threadIdx.y;

    if (i < ny) {
        for (int k = 0; k < BLOCK_X; k+= BLOCK_Y) {
            if ((j + k) >= nx) {
                break;
            }
            tr_pop[(j + k)*ny + i] = tile[threadIdx.x][threadIdx.y + k];
        }
    }

}

__global__ void init_int_array(int value, int n, int *array) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        array[i] = value;
    }
}

__global__ void init_int_array_limited(int value, int n, int *array) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        array[i] = value;
    }
}

__global__ void find_positions(int n, int m, int *sorted_idx, int *positions, float *tr_pop) {     
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < n & j < m) {
        int idx = sorted_idx[j*n + i];
        float my_val = tr_pop[j*n + i];
        int next = 1;
        while (i + next < n && my_val == tr_pop[j*n + i + next]) {
            next++;
        }
        positions[j*n + idx] = i + ((next > 1) * next);
    }
}

__global__ void best_objective(int n, int m, int *positions, int *best_m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int min_pos = positions[i];
        int obj = 0;
        for (int j = 1; j < m; j++) {
            int pos = positions[j*n + i];
            if (pos < min_pos) {
                min_pos = pos;
                obj = j;
            }
        }
        best_m[i] = min_pos;
        best_m[n + i] = obj;
    }
}

__device__ int is_dominated(int n, int m, int a, int b, const float *tr_pop) {
    bool equal = true;
    for (int j = 0; j < m; j++) {
        if (tr_pop[j*n + a] < tr_pop[j*n + b]) {
            return 0;
        }
        equal = equal & !(tr_pop[j*n + a] > tr_pop[j*n + b]);
    }
    return !equal;
}

__device__ void domination_check_inner(int n, int m, int idx, const float *tr_pop,
                                 const int *sorted_idx, const int *best_m,
                                 int *dominating_idx, int *last_batch,
                                 int *final_ranks, int curr_rank) {

    __shared__  bool blk_dominated;

    if (final_ranks[idx] >= 0) {
        return;
    }
    if (threadIdx.x == 0) {
        blk_dominated = false;
    }

    __syncthreads();
    
    const int i = idx*blockDim.x + threadIdx.x;
    int dom_idx = dominating_idx[i];
    if (dom_idx >= 0) {
        int dom_rank = final_ranks[dom_idx];
        if (dom_rank >= 0 && dom_rank < curr_rank) {
            dominating_idx[i] = -1;
        } else {
            blk_dominated = true;
        }
    }

    __syncthreads();

    if (blk_dominated) {
        return;
    }

    const int pos = best_m[idx];
    const int offset = best_m[n + idx]*n;
    for (int j = last_batch[idx]; j < pos; j += blockDim.x) {

        __syncthreads();

        if (j + threadIdx.x < pos) {
            int j_idx = sorted_idx[offset + j + threadIdx.x];

            bool is_candidate = true;
            if (curr_rank > 0) {
                int j_rank = final_ranks[j_idx];
                if (j_rank >= 0 && j_rank < curr_rank) {
                    is_candidate = false;
                }
            }
            if (is_candidate && is_dominated(n, m, idx, j_idx, tr_pop)) {
                dominating_idx[i] = j_idx;
                blk_dominated = true;
            }
        }

        __syncthreads();

        if (blk_dominated) {
            if (threadIdx.x == 0) {
                last_batch[idx] = j + blockDim.x;
            }
            return;
        }
    }

    if (threadIdx.x == 0) {
        final_ranks[idx] = curr_rank;
    }
}

__global__ void domination_check(int n, int m, const float *tr_pop,
                                 const int *sorted_idx, const int *best_m,
                                 int *dominating_idx, int *last_batch,
                                 int *final_ranks, int curr_rank) {

    for (int idx = blockIdx.x; idx < n; idx += gridDim.x) {
        domination_check_inner(n, m, idx, tr_pop, sorted_idx, best_m, dominating_idx, last_batch, final_ranks, curr_rank);
        __syncthreads();
    }
}

void nds(int n, int m, float *h_pop, int verbosity, int *h_ranks) {
    int count = n*m;

    // Transpose population matrix
    float *d_pop, *d_tr_pop;
    checkCuda(hipMalloc(&d_pop, count*sizeof(float)));
    checkCuda(hipMalloc(&d_tr_pop, count*sizeof(float)));
    checkCuda(hipMemcpy(d_pop, h_pop, count*sizeof(float), hipMemcpyHostToDevice));
    {
        dim3 block(BLOCK_X, BLOCK_Y);
        dim3 grid((m + block.x - 1)/block.x, 
                  (ceil(n*BLOCK_Y/(float)BLOCK_X) + block.y - 1)/block.y);
        transpose<<<grid, block>>>(m, n, d_tr_pop, d_pop);
        checkCuda(hipGetLastError());
    }
    checkCuda(hipFree(d_pop));

    // Initialize population indexes
    int *d_sorted_idx;
    checkCuda(hipMalloc(&d_sorted_idx, count*sizeof(int)));
    {
        dim3 block(BLOCK_1D);
        dim3 grid((count + block.x - 1)/block.x);
        init_indexes<<<grid, block>>>(n, m, d_sorted_idx);
        checkCuda(hipGetLastError());
    }

    float *d_tr_pop_out;
    checkCuda(hipMalloc(&d_tr_pop_out, count*sizeof(float)));
    int *d_sorted_idx_out;
    checkCuda(hipMalloc(&d_sorted_idx_out, count*sizeof(int)));
    {
        hipStream_t *streams = (hipStream_t *) malloc(m*sizeof(hipStream_t));
        for (int j = 0; j < m; j++) {
            checkCuda(hipStreamCreate(&streams[j]));
        }

        // Determine temporary device storage requirements
        char     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                     d_tr_pop, d_tr_pop_out, d_sorted_idx, d_sorted_idx_out, n);

        // Allocate temporary storage
        checkCuda(hipMalloc(&d_temp_storage, m*temp_storage_bytes));

        for (int j = 0; j < m; j++) {
            // Run sorting operation
            hipcub::DeviceRadixSort::SortPairs(&d_temp_storage[j*temp_storage_bytes], temp_storage_bytes,
                        &d_tr_pop[j*n], &d_tr_pop_out[j*n], &d_sorted_idx[j*n], &d_sorted_idx_out[j*n], n, 0, sizeof(float)*8, streams[j]);
        }
        checkCuda(hipFree(d_temp_storage));
        
        for (int j = 0; j < m; j++) {
            checkCuda(hipStreamSynchronize(streams[j]));
            checkCuda(hipStreamDestroy(streams[j]));
        }
        checkCuda(hipFree(d_sorted_idx));
    }

    d_sorted_idx = d_sorted_idx_out;

    // Find out in which list each individual is higher.
    int *d_best_m;
    checkCuda(hipMalloc(&d_best_m, 2*n*sizeof(int)));
    {
        dim3 block(BLOCK_1D);
        dim3 grid((count + block.x - 1)/block.x);
        init_int_array<<<grid, block>>>(n, 2*n, d_best_m);
        checkCuda(hipGetLastError());
    }
    int *d_positions;
    checkCuda(hipMalloc(&d_positions, count*sizeof(int)));
    {
        dim3 block(BLOCK_X, BLOCK_Y);
        dim3 grid((n + block.x - 1)/block.x, (m + block.y - 1)/block.y);
        find_positions<<<grid, block>>>(n, m, d_sorted_idx, d_positions, d_tr_pop_out);
        checkCuda(hipGetLastError());
    }
    {
        dim3 block(BLOCK_1D);
        dim3 grid((n + block.x - 1)/block.x);
        best_objective<<<grid, block>>>(n, m, d_positions, d_best_m);
        checkCuda(hipGetLastError());
    }
    checkCuda(hipFree(d_positions));
    checkCuda(hipFree(d_tr_pop_out));

    {
        int *d_dominating_idx, *d_last_batch, *d_final_ranks;
        checkCuda(hipMalloc(&d_dominating_idx, n*BLOCK_1D*sizeof(int)));
        checkCuda(hipMalloc(&d_last_batch,   n*sizeof(int)));
        checkCuda(hipMalloc(&d_final_ranks,  n*sizeof(int)));
        {
            dim3 block(BLOCK_1D);
            dim3 grid((n + block.x - 1)/block.x);
            init_int_array<<<grid, block>>>( 0, n, d_last_batch);
            init_int_array<<<grid, block>>>(-1, n, d_final_ranks);
        }
        {
            dim3 block(BLOCK_1D);
            dim3 grid(min(n, 65535));
            init_int_array_limited<<<grid, block>>>(-1, n*BLOCK_1D, d_dominating_idx);

            int num_sorted = 0;
            int rank = 0;
            while (num_sorted < n) {
                domination_check<<<grid, block>>>(n, m, d_tr_pop, d_sorted_idx, 
                        d_best_m, d_dominating_idx, d_last_batch, 
                        d_final_ranks, rank);
                checkCuda(hipMemcpy(h_ranks, d_final_ranks, 
                                     n*sizeof(int), hipMemcpyDeviceToHost));
                int front_count = 0;
                for (int i = 0; i < n; i++) {
                    if (h_ranks[i] == rank) {
                        front_count += 1;
                    }

                }
                if (verbosity > 0) {
                    printf("Front %d: %d elements.\n", rank + 1, front_count);
                    if (verbosity > 1) {
                        for (int i = 0; i < n; i++) {
                            if (h_ranks[i] == rank) {
                                printf("%d ", i);
                            }

                        }
                        printf("\n");
                    }
                }
                rank++;
                num_sorted += front_count;
            }
        }
        checkCuda(hipFree(d_dominating_idx));
        checkCuda(hipFree(d_last_batch));
        checkCuda(hipFree(d_final_ranks));
    }

    
    checkCuda(hipFree(d_tr_pop));
    checkCuda(hipFree(d_sorted_idx));
    checkCuda(hipFree(d_best_m));
}

void show_info(int n, int m, char *filename, int verbosity) {
    printf("Parameters for this run:\n");
    printf("    Population size:      %d\n", n);
    printf("    Number of objectives: %d\n", m);
    if (filename != NULL) {
        printf("    Population data file: %s\n", filename);
    }
    printf("    Verbosity:            %d\n\n", verbosity);
}

#ifdef MATLAB_MEX_FILE
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, const mxArray *prhs[]) {
    hipFree(0); // Trick to initalize CUDA context
    
    printf("GPU-BOS: CUDA implementation of the Best Order Sort algorithm\n\n");
    int verbosity = 1;
    if (nrhs < 1) {
        mexErrMsgIdAndTxt("nds:nrhs", "Required input: Population matrix.");
    } 
    if (nlhs != 1) {
        mexErrMsgIdAndTxt("nds:nrhs", "Required output: Ranks array.");
    }
    if (!mxIsSingle(prhs[0]) || mxIsComplex(prhs[0])) {
        mexErrMsgIdAndTxt("nds:population", "Input population must be a single precision matrix.");
    }
    if (nrhs > 1) {
        if (!mxIsScalar(prhs[1])) {
            mexErrMsgIdAndTxt("nds:verbosity", "Verbosity must be a scalar.");
        }
        verbosity = mxGetScalar(prhs[1]);
        if (verbosity < 0 || verbosity > 2) {
            mexErrMsgIdAndTxt("nds:verbosity_level", "Verbosity must be 0, 1 or 2.");
        }
    }
    
    int n = mxGetN(prhs[0]);
    int m = mxGetM(prhs[0]);
    float *population = (float *) mxGetPr(prhs[0]);
    show_info(n, m, NULL, verbosity);

    double start_time = get_time();
    plhs[0] = mxCreateNumericMatrix(1, (mwSize)n, mxINT32_CLASS, mxREAL);
    int *ranks = (int *) mxGetPr(plhs[0]);
    nds(n, m, population, verbosity, ranks);
    double end_time = get_time();
    if (verbosity == 0) {
        printf("Elapsed time: %.9f ms.\n", end_time - start_time);
    }
}
#else
int main(int argc, char **argv) {
    hipSetDeviceFlags(hipDeviceMapHost);
    hipFree(0); // Trick to initalize CUDA context

    printf("GPU-BOS: CUDA implementation of the Best Order Sort algorithm\n\n");
    int verbosity = 1;

    int c, error;
    while ((c = getopt(argc, argv, "hv:")) != -1) {
        switch (c) {
            case 'h':
                send_help(argv[0]);
                exit(EXIT_SUCCESS);
            case 'v':
                error = parse_int(optarg, &verbosity);
                if (error) {
                    fprintf(stderr, "ERROR (-v): Invalid verbosity level.\n");
                    exit(EXIT_FAILURE);
                }
                break;
            default:
                send_help(argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    int n_opts = argc - optind;
    if (!n_opts) {
        send_help(argv[0]);
        exit(EXIT_FAILURE);
    }
    if (n_opts < 3) {
        fprintf(stderr, "ERROR: Missing required parameters.\n");
        exit(EXIT_FAILURE);
    }
    if (n_opts > 3) {
        fprintf(stderr, "WARNING: Too many non-optional arguments!\n");
    } 

    int n, m;
    error = parse_int(argv[optind++], &n);
    if (error) {
        fprintf(stderr, "ERROR (n): Invalid population size.\n");
        exit(EXIT_FAILURE);
    }

    error = parse_int(argv[optind++], &m);
    if (error) {
        fprintf(stderr, "ERROR (m): Invalid number of objectives.\n");
        exit(EXIT_FAILURE);
    }

    char filename[BUFSIZE];
    strcpy(filename, argv[optind++]);
    FILE *f = fopen(filename, "r");
    
    if (f == NULL) {
        fprintf(stderr, "ERROR (pop_file): Unable to open population file.\n");
        exit(EXIT_FAILURE);
    }

    // Read population data
    float *h_pop = (float *) malloc(n*m*sizeof(float));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            if(!fscanf(f, "%f", &h_pop[i*m + j])) {
                fprintf(stderr, "ERROR: While reading population.");
                exit(EXIT_FAILURE);
            }
        }
    }
    fclose(f);

    show_info(n, m, filename, verbosity);

    double start_time = get_time();
    int *h_ranks = (int *) malloc(n*sizeof(int));
    nds(n, m, h_pop, verbosity, h_ranks);
    free(h_ranks);
    double end_time = get_time();
    if (verbosity == 0) {
        printf("Elapsed time: %.9f ms.\n", end_time - start_time);
    }

    free(h_pop);
	exit(EXIT_SUCCESS);
}
#endif
